#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cmath>
#include <climits>
#include <time.h>
#include <new>
#include <algorithm>

__constant__ int EdgeTable[8];
__constant__ int VertexTable[6];

struct ResultData
{
	int thread_id;
	int color_index;
};

void Initialize_graph(int*& e_t, int *& v_t, int & n, int & e)
{
	e = 8;
	e_t = new int[e];
	e_t[0] = 5;
	e_t[1] = 5;
	e_t[2] = 4;
	e_t[3] = 3;
	e_t[4] = 5;
	e_t[5] = 1;
	e_t[6] = 2;
	e_t[7] = 4;

	n = 5;
	v_t = new int[n+1];
	v_t[0] = 0;
	v_t[1] = 1;
	v_t[2] = 2;
	v_t[3] = 3;
	v_t[4] = 5;
	v_t[5] = 8;
}

__device__ bool TestColoriage(int coloriage[], int size_v)
{
	bool result = true;
	int current_vertex = 0;
	while (result && current_vertex < size_v)
	{
		//get edges range
		int start_edge_index = VertexTable[current_vertex];
		int end_edge_index = VertexTable[current_vertex+1];

		for (int i = start_edge_index; i < end_edge_index; i++)
		{
			if (coloriage[current_vertex] == coloriage[EdgeTable[i]])
			{
				result = false;
				break;
			}
		}
		current_vertex += 1;
	}
	return result;
}
__device__ int dev_pow(int a, int b)
{
	uint64_t result = 1;
	while (b-- > 0)
	{
		result *= a;
	}
	return result;
}

__global__ void BruteForceKernel(const int *n, int *output, bool * found_flag)
{
	int *colors = new int[*n];
	//int tid = threadIdx.x;
	//if (tid == 0)
	//{
	//	for (int i = 0; i < *n; i++)
	//		output[i] = i;
	//	*found_flag = true;
	//}

	uint64_t blockId_grid = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
	uint64_t threads_per_block = blockDim.x*blockDim.y*blockDim.z;
	//numer w�tku
	uint64_t tid = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y + blockId_grid * threads_per_block;
	//liczba kolorowa�
	uint64_t all_results = dev_pow(3, *n);
	//dzielimy przedzia� kolorowa� (funkcja numeru w�tku i ilo�ci w�tk�w) - ka�dy pe�ny, ostatni resztki z dzielenia
	uint64_t threads = blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z;
	uint64_t colors_count = (uint64_t)(all_results / threads + 1);
	uint64_t start_key = tid * colors_count;
	uint64_t end_key = start_key + colors_count;

	if (tid == 0)
	{
		output = new int[*n];
		for (int i = 0; i < *n; i++)
			output[i] = -2;
	}

	if (start_key >= all_results)
		return;
	if (end_key >= all_results)
		end_key = all_results;

	while (!(*found_flag) && start_key < end_key)
	{
		uint64_t tmp_key = start_key;
		//mapowanie klucza na kolorowanie (system tr�jkowy)
		for (int i = 0; i < *n; i++)
		{
			colors[i] = tmp_key % 3;
			tmp_key = tmp_key / 3;
		}
		//testowanie kolorowania
		//ob�o�y� og�lnocudowym mutexem
		if (TestColoriage(colors, *n) && !*found_flag)
		{
			*found_flag = true;
			for (int i = 0; i < *n; i++)
				output[i] = colors[i];
		}
		//kolejny klucz
		start_key += 1;
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t ColorWithCuda(int e_t[], int v_t[], int *n, int *e, bool * flag, int* output)
{
	int *dev_e_t, *dev_v_t, *dev_n;
	int *dev_output;
	bool *dev_flag;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors and two values (four input, one output).
	cudaStatus = hipMalloc((void**)&dev_e_t, (*e) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_v_t, (*n) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_n, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_output, (*n) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_flag, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU const memory.
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(EdgeTable), e_t, (*e) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(VertexTable), v_t, (*n + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_n, n, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_flag, flag, sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	BruteForceKernel << <1, 8 >> > (dev_n, dev_output, dev_flag);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, dev_output, (*n) * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(flag, dev_flag, sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_e_t);
	hipFree(dev_v_t);
	hipFree(dev_n);
	hipFree(dev_flag);
	hipFree(dev_output);

	return cudaStatus;
}

int main()
{
	int *edge_table, *vertex_table;
	int n, e;
	int *output;
	bool flag = false;

	//read and translate graph
	Initialize_graph(edge_table, vertex_table, n, e);

	output = new int[n];
	std::fill_n(output, n, -1);

	for (int i = 0; i < n; i++)
		printf("%d -> %d\n", i, output[i]);

    // Add vectors in parallel.
    hipError_t cudaStatus = ColorWithCuda(edge_table, vertex_table, &n, &e, &flag, output);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ColorWithCuda failed! %d", cudaStatus);
        return 1;
    }

	printf("Done. Found = %s\n", flag?"true":"false");

	for (int i = 0; i < n; i++)
		printf("%d -> %d\n", i, output[i]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	delete[] edge_table;
	delete[] vertex_table;
	delete[] output;

    return 0;
}
